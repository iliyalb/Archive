#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void vectorAdd(int *a, int *b, int *c)
{
    // A list of thread called for every index
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    // Locally allocated data on device
    int a[] = {1, 2, 3};
    int b[] = {4, 5, 6};
    int c[sizeof(a) / sizeof(int)] = {0};

    // Declare pointers, allocate GPU memory and copy the vectors
    int *cudaA = 0, cudaB = 0, cudaC = 0;
    hipMalloc(&cudaA, sizeof(a));
    hipMalloc(&cudaB, sizeof(b));
    hipMalloc(&cudaC, sizeof(c));
    hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);

    // Run sizeof many times in parallel and copy out of the GPU
    vectorAdd<<<1, sizeof(a) / sizeof(int)>>>(cudaA, cudaB, cudaC);
    hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);
    return 0;
}
